#include "utils.h"
#include "cuda.cuh"
#include "check.cuh"


void FreeSDM3D(SDM3D* x)
{
    delete[] x->xx;
    delete[] x->yy;
    delete[] x->zz;
    delete[] x->xy;
    delete[] x->xz;
    delete[] x->yz;
    delete x;
}

// 只是把hessian3D的子元素分配显存
void CudaMallocSDM3D(SDM3D* item, int imageSize)
{
    CUDA_CHECK(hipMalloc((void**)&item->xx, sizeof(float) * imageSize));
    CUDA_CHECK(hipMalloc((void**)&item->yy, sizeof(float) * imageSize));
    CUDA_CHECK(hipMalloc((void**)&item->zz, sizeof(float) * imageSize));
    CUDA_CHECK(hipMalloc((void**)&item->xy, sizeof(float) * imageSize));
    CUDA_CHECK(hipMalloc((void**)&item->xz, sizeof(float) * imageSize));
    CUDA_CHECK(hipMalloc((void**)&item->yz, sizeof(float) * imageSize));
}
void CudaFreeSDM3D(SDM3D* item)
{
    CUDA_CHECK(hipFree(item->xx));
    CUDA_CHECK(hipFree(item->yy));
    CUDA_CHECK(hipFree(item->zz));
    CUDA_CHECK(hipFree(item->xy));
    CUDA_CHECK(hipFree(item->xz));
    CUDA_CHECK(hipFree(item->yz));
    delete item;
}












