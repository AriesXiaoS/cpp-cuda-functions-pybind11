#include "hip/hip_runtime.h"
#include "common.h"
#include "define.h"
#include "utils.h"
#include "cuda.cuh"
#include "check.cuh"

#include <chrono>   
using namespace chrono;


void printDuration(std::chrono::time_point<std::chrono::system_clock> start, 
 std::string msg){
    std::chrono::time_point<std::chrono::system_clock> end;
    end = std::chrono::system_clock::now();
    std::chrono::duration<double> elapsed_seconds = end-start;
    std::cout << msg << " time: " << elapsed_seconds.count() << "s\n";
}

// 卷积高斯二阶导 得到 Hessian矩阵
template <typename T>
void ConvHessian(T* paddedImage_d, SDM3D<T>* kernels, SDM3D<T>* hessian,
                 int* imgShape, int kernelSize, dim3 dimBlock, dim3 dimGrid)
{
    int kernelSizeFlattened = kernelSize * kernelSize * kernelSize;
    // kernel on device
    SDM3D<T>* kernels_d = new SDM3D<T>();
    CudaMallocSDM3D(kernels_d, kernelSizeFlattened);
    Conv3DParam convParams = {int(kernelSize), 
    { int(imgShape[0]), int(imgShape[1]), int(imgShape[2]), } };
    // xx
    CUDA_CHECK(hipMemcpy(kernels_d->xx, kernels->xx, sizeof(T) * kernelSizeFlattened, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(kernels_d->yy, kernels->yy, sizeof(T) * kernelSizeFlattened, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(kernels_d->zz, kernels->zz, sizeof(T) * kernelSizeFlattened, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(kernels_d->xy, kernels->xy, sizeof(T) * kernelSizeFlattened, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(kernels_d->xz, kernels->xz, sizeof(T) * kernelSizeFlattened, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(kernels_d->yz, kernels->yz, sizeof(T) * kernelSizeFlattened, hipMemcpyHostToDevice));

    CudaConv3D<<<dimGrid, dimBlock>>>(paddedImage_d, hessian->xx, 
                                        kernels_d->xx, convParams);
    // yy
    CudaConv3D<<<dimGrid, dimBlock>>>(paddedImage_d, hessian->yy, 
                                        kernels_d->yy, convParams);
    // zz
    CudaConv3D<<<dimGrid, dimBlock>>>(paddedImage_d, hessian->zz, 
                                        kernels_d->zz, convParams);
    // xy
    CudaConv3D<<<dimGrid, dimBlock>>>(paddedImage_d, hessian->xy, 
                                        kernels_d->xy, convParams);
    // xz
    CudaConv3D<<<dimGrid, dimBlock>>>(paddedImage_d, hessian->xz, 
                                        kernels_d->xz, convParams);
    // yz
    CudaConv3D<<<dimGrid, dimBlock>>>(paddedImage_d, hessian->yz, 
                                        kernels_d->yz, convParams);
    //
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());

    CudaFreeSDM3D(kernels_d);
    delete kernels_d;
}

template <typename T>
T* VoMax(T** outputs, int sigmaLen, int imageSize)
{
    T* maxOutput = new T[imageSize];
    for(int i=0; i<imageSize; i++)
    {
        maxOutput[i] = outputs[0][i];
        for(int j=1; j<sigmaLen; j++)
        {
            if(outputs[j][i] > maxOutput[i])
            {
                maxOutput[i] = outputs[j][i];
            }
        }
    }
    return maxOutput;
}



template <typename T>
map<string, py::array_t<T>> CudaFrangi3D(
    py::array_t<T> image, int device, std::vector<T> sigmas,
    T alpha, T beta, T gamma, bool blackRidges,
    int maxIters, T tolerance, int eigenVectorType,
    int verbose, std::vector<int> cudaDimBlock)
{
    if(verbose >= 1){
        printf("Data type size: %d\n", int(sizeof(T)));
    }
    ////
    std::chrono::time_point<std::chrono::system_clock> start;
    auto img = image.request();
    
    if(img.ndim != 3) throw std::runtime_error("Number of dimensions must be 3");
    if(device<0) throw std::runtime_error("Device number must be non-negative");
    if(alpha<=0 || alpha >1) throw std::runtime_error("Alpha must be in (0, 1]");
    if(beta<=0 || beta >1) throw std::runtime_error("Beta must be in (0, 1]");
    if(gamma<0) throw std::runtime_error("Gamma must be non-negative");
    if(maxIters<=0) throw std::runtime_error("MaxIters must be positive");
    if(tolerance<0) throw std::runtime_error("Tolerance must be non-negative");
    if(eigenVectorType<VEC_TYPE_NONE || eigenVectorType>VEC_TYPE_SPHERE) throw std::runtime_error("EigenVectorType must be 0, 1 or 2");
    for(int i=0; i<sigmas.size(); i++)
    {
        if(sigmas[i]<0) throw std::runtime_error("Sigma must be non-negative");
    }
    if(cudaDimBlock.size() != 3) throw std::runtime_error("cudaDimBlock must be a list of 3 integers");
    for(int i=0; i<3; i++)
    {
        if(cudaDimBlock[i]<=0) throw std::runtime_error("cudaDimBlock must be positive");
    }

    hipSetDevice(device);
    CUDA_CHECK(hipGetLastError());
    //
    T* ptr = (T*) img.ptr;
    int* imgShape = new int[3]{int(img.shape[0]), int(img.shape[1]), int(img.shape[2])};
    int imageSize = imgShape[0] * imgShape[1] * imgShape[2];
    if(!blackRidges){
        for(int i=0; i<imageSize; i++){
            ptr[i] = -ptr[i];
        }
    }
    
    // hessian 子元素在device 本体在host
    // hessian_d 本体在device    
    SDM3D<T>* hessian = new SDM3D<T>(); // 卷积后的
    SDM3D<T>* hessian_d;   
    CudaMallocSDM3D<T>(hessian, imageSize); 
    CUDA_CHECK(hipMalloc((void**)&hessian_d, sizeof(SDM3D<T>)));
    CUDA_CHECK(hipMemcpy(hessian_d, hessian, sizeof(SDM3D<T>), hipMemcpyHostToDevice));
    //
    Eigen3D<T>* eigen = new Eigen3D<T>();
    Eigen3D<T>* eigen_d;
    CUDA_CHECK(hipMalloc((void**)&eigen->eigenValues, sizeof(T) *3 *imageSize));
    if(eigenVectorType != VEC_TYPE_NONE){
        CUDA_CHECK(hipMalloc((void**)&eigen->eigenVectors, sizeof(T) *9 *imageSize));
    }
    CUDA_CHECK(hipMalloc((void**)&eigen_d, sizeof(Eigen3D<T>)));
    CUDA_CHECK(hipMemcpy(eigen_d, eigen, sizeof(Eigen3D<T>), hipMemcpyHostToDevice));
    //
    T* HFnorm = new T[imageSize];
    T* HFnorm_d;
    CUDA_CHECK(hipMalloc((void**)&HFnorm_d, sizeof(T) *imageSize));
    //
    T** outputs = new T*[sigmas.size()];
    for(int i=0; i<sigmas.size(); i++)
    {
        outputs[i] = new T[imageSize];
    }
    T* output_d;
    CUDA_CHECK(hipMalloc((void**)&output_d, sizeof(T) * imageSize));


    for(int i=0; i<sigmas.size(); i++)
    {
        start = std::chrono::system_clock::now();
        // 3 sigma 原则
        int kernelSize = 2 * ceil(3 * sigmas[i]) + 1;
        SDM3D<T>* kernels = GetGaussianKernels<T>(sigmas[i], kernelSize);
        // padding
        int paddingSize = kernelSize / 2;
        int paddedImageSize = (imgShape[0] + 2 * paddingSize) * (imgShape[1] + 2 * paddingSize) * (imgShape[2] + 2 * paddingSize);
        T* paddedImage = new T[paddedImageSize];
        PaddingFlattenedArr_3D<T>(ptr, paddedImage, 
                                imgShape[0], imgShape[1], imgShape[2],
                                0, paddingSize, paddingSize, paddingSize);
        // image to device
        T* paddedImage_d;
        CUDA_CHECK(hipMalloc((void**)&paddedImage_d, sizeof(T) * paddedImageSize));
        CUDA_CHECK(hipMemcpy(paddedImage_d, paddedImage, sizeof(T) * paddedImageSize, hipMemcpyHostToDevice));
        // Conv Hessian
        
        dim3 dimBlock(cudaDimBlock[0], cudaDimBlock[1], cudaDimBlock[2]);
        dim3 dimGrid( ceil( (imgShape[0] + 2 * paddingSize) / dimBlock.x) + 1,
                        ceil( (imgShape[1] + 2 * paddingSize) / dimBlock.y) + 1,
                        ceil( (imgShape[2] + 2 * paddingSize) / dimBlock.z) + 1 );
        
        ConvHessian<T>(paddedImage_d, kernels, hessian, imgShape, kernelSize, dimBlock, dimGrid);
        
        CUDA_CHECK(hipMemcpy(hessian_d, hessian, sizeof(SDM3D<T>), hipMemcpyHostToDevice));
        // Hessian Eigen 重点耗时
        CudaHessianEigen<T><<<dimGrid, dimBlock>>>(hessian_d, eigen_d, HFnorm_d,
            imgShape[0], imgShape[1], imgShape[2], maxIters, tolerance, eigenVectorType);        
        CUDA_CHECK(hipGetLastError());
        CUDA_CHECK(hipDeviceSynchronize());
        // printf("CudaHessianEigen Done\n");
        T frangi_c;
        if(gamma <= 0){
            CUDA_CHECK(hipMemcpy(HFnorm, HFnorm_d, sizeof(T) * imageSize, hipMemcpyDeviceToHost));
            frangi_c = *std::max_element(HFnorm, HFnorm + imageSize) * 0.5;
        }else{
            frangi_c = gamma;
        }

        CudaFrangiVo<T><<<dimGrid, dimBlock>>>(eigen_d, output_d,
            imgShape[0], imgShape[1], imgShape[2], alpha, beta, frangi_c, blackRidges);
        CUDA_CHECK(hipGetLastError());
        CUDA_CHECK(hipDeviceSynchronize());
        
        CUDA_CHECK(hipMemcpy(outputs[i], output_d, sizeof(T) * imageSize, hipMemcpyDeviceToHost));
        // free device
        CUDA_CHECK(hipFree(paddedImage_d));
        // free host
        delete[] paddedImage;
        FreeSDM3D<T>(kernels);
        
        if(verbose >= 1){
            printf("%d/%d sigma %f done - ", i+1, int(sigmas.size()), sigmas[i]);
            printDuration(start, "iter");
        }

    }

    // free device
    CUDA_CHECK(hipFree(output_d));
    CUDA_CHECK(hipFree(HFnorm_d));
    CudaFreeSDM3D(hessian);
    CUDA_CHECK(hipFree(hessian_d));
    CUDA_CHECK(hipFree(eigen->eigenValues));
    if(eigenVectorType != VEC_TYPE_NONE){
        CUDA_CHECK(hipFree(eigen->eigenVectors));
    }
    CUDA_CHECK(hipFree(eigen_d));


    T* frangi = VoMax<T>(outputs, sigmas.size(), imageSize);

    auto frangi_pyArr = py::array_t<T>(imageSize, frangi);
    frangi_pyArr.resize({imgShape[0], imgShape[1], imgShape[2]});

    map<string, py::array_t<T>> result;
    result["frangi"] = frangi_pyArr;

    return result;

}



template map<string, py::array_t<float>> CudaFrangi3D<float>(
    py::array_t<float> image, int device, std::vector<float> sigmas,
    float alpha, float beta, float gamma, bool blackRidges,
    int maxIters, float tolerance, int eigenVectorType,
    int verbose, std::vector<int> cudaDimBlock);
template map<string, py::array_t<double>> CudaFrangi3D<double>(
    py::array_t<double> image, int device, std::vector<double> sigmas,
    double alpha, double beta, double gamma, bool blackRidges,
    int maxIters, double tolerance, int eigenVectorType,
    int verbose, std::vector<int> cudaDimBlock);





















