#include "common.h"
#include "utils.h"
#include "define.h"

#include "cuda.cuh"
#include "check.cuh"



map<string, py::array_t<float>> HessianEigenTest_3x3(
    py::array_t<float> A, int vecType,
    int device, int maxIters, float tolerance)
{
    auto buf = A.request();
    float* ptr = (float*) buf.ptr;
    if(buf.ndim != 2){
        throw std::runtime_error("Number of dimensions must be 2");
    }
    if(buf.shape[0] != 3 || buf.shape[1] != 3){
        throw std::runtime_error("Matrix must be 3x3");
    }
    hipSetDevice(device);
    CUDA_CHECK(hipGetLastError());

    Hessian3D* hessian = new Hessian3D();
    Hessian3D* hessian_d;    
    CUDA_CHECK(hipMalloc((void**)&hessian->Ixx, sizeof(float) * 2));
    CUDA_CHECK(hipMalloc((void**)&hessian->Iyy, sizeof(float) * 2));
    CUDA_CHECK(hipMalloc((void**)&hessian->Izz, sizeof(float) * 2));
    CUDA_CHECK(hipMalloc((void**)&hessian->Ixy, sizeof(float) * 2));
    CUDA_CHECK(hipMalloc((void**)&hessian->Ixz, sizeof(float) * 2));
    CUDA_CHECK(hipMalloc((void**)&hessian->Iyz, sizeof(float) * 2));
    CUDA_CHECK(hipMemcpy(hessian->Ixx, ptr, sizeof(float) * 2, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(hessian->Iyy, ptr + 4, sizeof(float) * 2, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(hessian->Izz, ptr + 8, sizeof(float) * 2, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(hessian->Ixy, ptr + 1, sizeof(float) * 2, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(hessian->Ixz, ptr + 2, sizeof(float) * 2, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(hessian->Iyz, ptr + 5, sizeof(float) * 2, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMalloc((void**)&hessian_d, sizeof(Hessian3D)));
    CUDA_CHECK(hipMemcpy(hessian_d, hessian, sizeof(Hessian3D), hipMemcpyHostToDevice));

    Eigen3D* eigen = new Eigen3D();
    Eigen3D* eigen_d;
    CUDA_CHECK(hipMalloc((void**)&eigen->eigenValues, sizeof(float) * 3));
    CUDA_CHECK(hipMalloc((void**)&eigen->eigenVectors, sizeof(float) * 9));
    CUDA_CHECK(hipMalloc((void**)&eigen_d, sizeof(Eigen3D)));
    CUDA_CHECK(hipMemcpy(eigen_d, eigen, sizeof(Eigen3D), hipMemcpyHostToDevice));

    float* HFnorm_d;
    CUDA_CHECK(hipMalloc((void**)&HFnorm_d, sizeof(float) * 3));

    int* imgShape = new int[3]{1, 2, 1};
    //
    dim3 dimBlock(1, 1, 1);
    dim3 dimGrid(1, 1, 1);
    SetHessianParams<<<dimGrid, dimBlock>>>(imgShape[0],  imgShape[1], imgShape[2], 
                                            maxIters, tolerance, vecType);
    CudaHessianEigen<<<dimGrid, dimBlock>>>(hessian_d, eigen_d, HFnorm_d);
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());

    auto eigenValue_pyArray = py::array_t<float>(3);
    auto eigenValue_buf = eigenValue_pyArray.request();
    float* eigenValue_ptr = (float*) eigenValue_buf.ptr;

    auto eigenVector_pyArray = py::array_t<float>(9);
    auto eigenVector_buf = eigenVector_pyArray.request();
    float* eigenVector_ptr = (float*) eigenVector_buf.ptr;

    CUDA_CHECK(hipMemcpy(eigen, eigen_d, sizeof(Eigen3D), hipMemcpyDeviceToHost));

    CUDA_CHECK(hipMemcpy(eigenValue_ptr, eigen->eigenValues, sizeof(float) * 3, hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(eigenVector_ptr, eigen->eigenVectors, sizeof(float) * 9, hipMemcpyDeviceToHost));

    eigenVector_pyArray.resize({3, 3});
    map<string, py::array_t<float>> result;
    result["eigenValues"] = eigenValue_pyArray;
    result["eigenVectors"] = eigenVector_pyArray;

    // 释放内存
    CUDA_CHECK(hipFree(hessian->Ixx));
    CUDA_CHECK(hipFree(hessian->Iyy));
    CUDA_CHECK(hipFree(hessian->Izz));
    CUDA_CHECK(hipFree(hessian->Ixy));
    CUDA_CHECK(hipFree(hessian->Ixz));
    CUDA_CHECK(hipFree(hessian->Iyz));

    CUDA_CHECK(hipFree(eigen->eigenValues));
    CUDA_CHECK(hipFree(eigen->eigenVectors));

    delete [] imgShape;

    return result;

}   





