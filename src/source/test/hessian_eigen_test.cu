#include "common.h"
#include "utils.h"
#include "define.h"

#include "cuda.cuh"
#include "check.cuh"



map<string, py::array_t<float>> HessianEigenTest_3x3(
    py::array_t<float> A, int vecType,
    int device, int maxIters, float tolerance)
{
    auto buf = A.request();
    float* ptr = (float*) buf.ptr;
    if(buf.ndim != 2){
        throw std::runtime_error("Number of dimensions must be 2");
    }
    if(buf.shape[0] != 3 || buf.shape[1] != 3){
        throw std::runtime_error("Matrix must be 3x3");
    }
    hipSetDevice(device);
    CUDA_CHECK(hipGetLastError());

    SDM3D* hessian = new SDM3D();
    SDM3D* hessian_d;    
    CUDA_CHECK(hipMalloc((void**)&hessian->xx, sizeof(float) * 2));
    CUDA_CHECK(hipMalloc((void**)&hessian->yy, sizeof(float) * 2));
    CUDA_CHECK(hipMalloc((void**)&hessian->zz, sizeof(float) * 2));
    CUDA_CHECK(hipMalloc((void**)&hessian->xy, sizeof(float) * 2));
    CUDA_CHECK(hipMalloc((void**)&hessian->xz, sizeof(float) * 2));
    CUDA_CHECK(hipMalloc((void**)&hessian->yz, sizeof(float) * 2));
    CUDA_CHECK(hipMemcpy(hessian->xx, ptr, sizeof(float) * 2, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(hessian->yy, ptr + 4, sizeof(float) * 2, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(hessian->zz, ptr + 8, sizeof(float) * 2, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(hessian->xy, ptr + 1, sizeof(float) * 2, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(hessian->xz, ptr + 2, sizeof(float) * 2, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(hessian->yz, ptr + 5, sizeof(float) * 2, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMalloc((void**)&hessian_d, sizeof(SDM3D)));
    CUDA_CHECK(hipMemcpy(hessian_d, hessian, sizeof(SDM3D), hipMemcpyHostToDevice));

    Eigen3D* eigen = new Eigen3D();
    Eigen3D* eigen_d;
    CUDA_CHECK(hipMalloc((void**)&eigen->eigenValues, sizeof(float) * 3));
    CUDA_CHECK(hipMalloc((void**)&eigen->eigenVectors, sizeof(float) * 9));
    CUDA_CHECK(hipMalloc((void**)&eigen_d, sizeof(Eigen3D)));
    CUDA_CHECK(hipMemcpy(eigen_d, eigen, sizeof(Eigen3D), hipMemcpyHostToDevice));

    float* HFnorm_d;
    CUDA_CHECK(hipMalloc((void**)&HFnorm_d, sizeof(float) * 3));

    int* imgShape = new int[3]{1, 2, 1};
    //
    dim3 dimBlock(1, 1, 1);
    dim3 dimGrid(1, 1, 1);
    SetHessianParams<<<dimGrid, dimBlock>>>(imgShape[0],  imgShape[1], imgShape[2], 
                                            maxIters, tolerance, vecType);
    CudaHessianEigen<<<dimGrid, dimBlock>>>(hessian_d, eigen_d, HFnorm_d);
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());

    auto eigenValue_pyArray = py::array_t<float>(3);
    auto eigenValue_buf = eigenValue_pyArray.request();
    float* eigenValue_ptr = (float*) eigenValue_buf.ptr;

    auto eigenVector_pyArray = py::array_t<float>(9);
    auto eigenVector_buf = eigenVector_pyArray.request();
    float* eigenVector_ptr = (float*) eigenVector_buf.ptr;

    CUDA_CHECK(hipMemcpy(eigen, eigen_d, sizeof(Eigen3D), hipMemcpyDeviceToHost));

    CUDA_CHECK(hipMemcpy(eigenValue_ptr, eigen->eigenValues, sizeof(float) * 3, hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(eigenVector_ptr, eigen->eigenVectors, sizeof(float) * 9, hipMemcpyDeviceToHost));

    eigenVector_pyArray.resize({3, 3});
    map<string, py::array_t<float>> result;
    result["eigenValues"] = eigenValue_pyArray;
    result["eigenVectors"] = eigenVector_pyArray;

    // 释放内存
    CUDA_CHECK(hipFree(hessian->xx));
    CUDA_CHECK(hipFree(hessian->yy));
    CUDA_CHECK(hipFree(hessian->zz));
    CUDA_CHECK(hipFree(hessian->xy));
    CUDA_CHECK(hipFree(hessian->xz));
    CUDA_CHECK(hipFree(hessian->yz));

    CUDA_CHECK(hipFree(eigen->eigenValues));
    CUDA_CHECK(hipFree(eigen->eigenVectors));

    delete [] imgShape;

    return result;

}   





