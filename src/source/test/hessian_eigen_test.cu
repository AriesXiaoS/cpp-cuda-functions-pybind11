#include "common.h"
#include "utils.h"
#include "define.h"

#include "cuda.cuh"
#include "check.cuh"


template <typename T>
map<string, py::array_t<T>> HessianEigenTest_3x3(
    py::array_t<T> A, int vecType,
    int device, int maxIters, T tolerance)
{
    auto buf = A.request();
    T* ptr = (T*) buf.ptr;
    if(buf.ndim != 2){
        throw std::runtime_error("Number of dimensions must be 2");
    }
    if(buf.shape[0] != 3 || buf.shape[1] != 3){
        throw std::runtime_error("Matrix must be 3x3");
    }
    hipSetDevice(device);
    CUDA_CHECK(hipGetLastError());

    SDM3D<T>* hessian = new SDM3D<T>();
    SDM3D<T>* hessian_d;    
    CUDA_CHECK(hipMalloc((void**)&hessian->xx, sizeof(T) * 2));
    CUDA_CHECK(hipMalloc((void**)&hessian->yy, sizeof(T) * 2));
    CUDA_CHECK(hipMalloc((void**)&hessian->zz, sizeof(T) * 2));
    CUDA_CHECK(hipMalloc((void**)&hessian->xy, sizeof(T) * 2));
    CUDA_CHECK(hipMalloc((void**)&hessian->xz, sizeof(T) * 2));
    CUDA_CHECK(hipMalloc((void**)&hessian->yz, sizeof(T) * 2));
    CUDA_CHECK(hipMemcpy(hessian->xx, ptr, sizeof(T) * 2, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(hessian->yy, ptr + 4, sizeof(T) * 2, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(hessian->zz, ptr + 8, sizeof(T) * 2, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(hessian->xy, ptr + 1, sizeof(T) * 2, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(hessian->xz, ptr + 2, sizeof(T) * 2, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(hessian->yz, ptr + 5, sizeof(T) * 2, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMalloc((void**)&hessian_d, sizeof(SDM3D<T>)));
    CUDA_CHECK(hipMemcpy(hessian_d, hessian, sizeof(SDM3D<T>), hipMemcpyHostToDevice));

    Eigen3D<T>* eigen = new Eigen3D<T>();
    Eigen3D<T>* eigen_d;
    CUDA_CHECK(hipMalloc((void**)&eigen->eigenValues, sizeof(T) * 3));
    CUDA_CHECK(hipMalloc((void**)&eigen->eigenVectors, sizeof(T) * 9));
    CUDA_CHECK(hipMalloc((void**)&eigen_d, sizeof(Eigen3D<T>)));
    CUDA_CHECK(hipMemcpy(eigen_d, eigen, sizeof(Eigen3D<T>), hipMemcpyHostToDevice));

    T* HFnorm_d;
    CUDA_CHECK(hipMalloc((void**)&HFnorm_d, sizeof(T) * 3));

    int* imgShape = new int[3]{1, 2, 1};
    //
    dim3 dimBlock(1, 1, 1);
    dim3 dimGrid(1, 1, 1);
    // SetHessianParams<<<dimGrid, dimBlock>>>(imgShape[0],  imgShape[1], imgShape[2], 
    //                                         maxIters, tolerance, vecType);
    CudaHessianEigen<T><<<dimGrid, dimBlock>>>(hessian_d, eigen_d, HFnorm_d,
            imgShape[0], imgShape[1], imgShape[2], maxIters, tolerance, vecType);
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());

    auto eigenValue_pyArray = py::array_t<T>(3);
    auto eigenValue_buf = eigenValue_pyArray.request();
    T* eigenValue_ptr = (T*) eigenValue_buf.ptr;

    auto eigenVector_pyArray = py::array_t<T>(9);
    auto eigenVector_buf = eigenVector_pyArray.request();
    T* eigenVector_ptr = (T*) eigenVector_buf.ptr;

    CUDA_CHECK(hipMemcpy(eigen, eigen_d, sizeof(Eigen3D<T>), hipMemcpyDeviceToHost));

    CUDA_CHECK(hipMemcpy(eigenValue_ptr, eigen->eigenValues, sizeof(T) * 3, hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(eigenVector_ptr, eigen->eigenVectors, sizeof(T) * 9, hipMemcpyDeviceToHost));

    eigenVector_pyArray.resize({3, 3});
    map<string, py::array_t<T>> result;
    result["eigenValues"] = eigenValue_pyArray;
    result["eigenVectors"] = eigenVector_pyArray;

    // 释放内存
    CUDA_CHECK(hipFree(hessian->xx));
    CUDA_CHECK(hipFree(hessian->yy));
    CUDA_CHECK(hipFree(hessian->zz));
    CUDA_CHECK(hipFree(hessian->xy));
    CUDA_CHECK(hipFree(hessian->xz));
    CUDA_CHECK(hipFree(hessian->yz));

    CUDA_CHECK(hipFree(eigen->eigenValues));
    CUDA_CHECK(hipFree(eigen->eigenVectors));

    delete [] imgShape;

    return result;

}   

template map<string, py::array_t<float>> HessianEigenTest_3x3<float>(
    py::array_t<float> A, int vecType,
    int device, int maxIters, float tolerance);
template map<string, py::array_t<double>> HessianEigenTest_3x3<double>(
    py::array_t<double> A, int vecType,
    int device, int maxIters, double tolerance);



