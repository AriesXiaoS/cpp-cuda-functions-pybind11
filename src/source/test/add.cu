#include "hip/hip_runtime.h"
#include "common.h"

#include "cuda.cuh"
#include "check.cuh"




template <typename T>
__global__ void KernelAdd
(T *vec1, T *vec2, T* res, int num_elements){
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < num_elements) {
        res[idx] = vec1[idx] + vec2[idx];
    }
}

template <typename T>
void CudaAdd(T* vec1, T* vec2, T* res, int num_elements){
    T* cuda_vec1, *cuda_vec2, *cuda_res;
    int size_bytes = num_elements*sizeof(T);
    // 分配显存空间
    CUDA_CHECK(hipMalloc((void**)&cuda_vec1, size_bytes));
    CUDA_CHECK(hipMalloc((void**)&cuda_vec2, size_bytes));    
    CUDA_CHECK(hipMalloc((void**)&cuda_res, size_bytes));

    // host -> device
    CUDA_CHECK(hipMemcpy(cuda_vec1, vec1, size_bytes, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(cuda_vec2, vec2, size_bytes, hipMemcpyHostToDevice));

    // kernel
    dim3 dimBlock(1024, 1, 1);
    dim3 dimGrid(ceil((T)num_elements / dimBlock.x));
    KernelAdd<T><<<dimGrid, dimBlock>>>(cuda_vec1, cuda_vec2, cuda_res, num_elements);
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());

    // device -> host
    CUDA_CHECK(hipMemcpy(res, cuda_res, size_bytes, hipMemcpyDeviceToHost));
    
    // 释放显存空间
    CUDA_CHECK(hipFree(cuda_vec1));
    CUDA_CHECK(hipFree(cuda_vec2));
    CUDA_CHECK(hipFree(cuda_res));
}


template <typename T>
py::array_t<T> AddNp(py::array_t<T> vec1, py::array_t<T> vec2, int device)
{   
    // c++ 17
    // if constexpr (std::is_same<T, int>::value) {
    //     std::cout << "Type is int" << std::endl;
    // } else if constexpr (std::is_same<T, float>::value) {
    //     std::cout << "Type is float" << std::endl;
    // } else if constexpr (std::is_same<T, double>::value) {
    //     std::cout << "Type is double" << std::endl;
    // }
    //
    auto buf1 = vec1.request(), buf2 = vec2.request();
    // check dim
    if( buf1.ndim != buf2.ndim ){
        std::stringstream strstr;
        strstr << "ndim of vec1, vec2, res should be same, but got " << buf1.ndim << " " << buf2.ndim ;
        throw std::runtime_error(strstr.str());
    }
    // check shape
    if( buf1.size != buf2.size ){
        std::stringstream strstr;
        strstr << "size of vec1, vec2, res should be same, but got " << buf1.size << " " << buf2.size ;
        throw std::runtime_error(strstr.str());
    }
    //
    auto result = py::array_t<T>(buf1.size);
    py::buffer_info buf3 = result.request();

    //获取numpy.ndarray 数据指针
    T* ptr1 = (T*)buf1.ptr;
    T* ptr2 = (T*)buf2.ptr;
    T* ptr3 = (T*)buf3.ptr;

    if(device==-1){
        // use cpu
        //申请空间
        for (int i = 0; i < buf1.size; i++)
        {
            ptr3[i] = ptr1[i] + ptr2[i];
        }
    }else{        
        hipSetDevice(device);
        CUDA_CHECK(hipGetLastError());
        CudaAdd<T>(ptr1, ptr2, ptr3, buf1.size);
    }
    result.resize(buf1.shape);
    return result;
}


template py::array_t<float> AddNp<float>(py::array_t<float> vec1, py::array_t<float> vec2, int device);
template py::array_t<double> AddNp<double>(py::array_t<double> vec1, py::array_t<double> vec2, int device);








